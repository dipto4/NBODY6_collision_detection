#include "hip/hip_runtime.h"
//#include <iostream>
#include <cstdio>
// #include <cutil.h>
#ifdef CUDA_5
#  include <hip/hip_runtime_api.h>
#  define CUDA_SAFE_CALL checkCudaErrors
#else
#  include <cutil.h>
#endif
#include "cuda_pointer.h"
#include "cuda_share.h"
#define NTHREAD 128


struct Particle{
	float2 pos[3];
	float mass;
	float pad;

	Particle(double x[3], double m){
		pos[0] = float2_split(x[0]);
		pos[1] = float2_split(x[1]);
		pos[2] = float2_split(x[2]);
		mass = (float)m;
	}
	Particle(int){
		pos[0].x = pos[0].y = pos[1].x = pos[1].y = pos[2].x = pos[2].y = mass = pad = 0.f;
	}
	__device__ Particle() {}
};

__global__ void pot_kernel(int n, int istart, Particle *ptcl, float2 *phi){
	__shared__ Particle jpbuf[NTHREAD];
	int i = NTHREAD * blockIdx.x + threadIdx.x;
	Particle ip = ptcl[i+istart-1];
	float2 phii = make_float2(0.f, 0.f);
	for(int j=0; j<n; j+= NTHREAD){
		__syncthreads();
		jpbuf[threadIdx.x] = ptcl[j + threadIdx.x];
		__syncthreads();
#pragma unroll 4
		for(int jj=0; jj<NTHREAD; jj++){
			// if(j+jj == i) continue;
			Particle &jp = jpbuf[jj];
			float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
			float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
			float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y);
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
		phii = float2_regularize(phii);
	}
	phi[i] = phii;
}

extern "C"  void gpunb_devinit_(int *irank);

void gpupot(
        int *rank,
        int istart,
        int ni,
		int n,
		double m[],
		double x[][3],
		double pot[]){
	gpunb_devinit_(rank);

	double t0 = get_wtime();
	cudaPointer <float2> phi;
	cudaPointer <Particle> ptcl;
	int ng = NTHREAD * (ni/NTHREAD + (ni%NTHREAD ? 1 : 0));
	int ntg = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));
	if (ntg<ng+istart) ntg += NTHREAD;

	phi.allocate(ng);
	ptcl.allocate(ntg);

    //    std::cout << n << " " << ng << " "<< ntg << std::endl;
	for(int i=0; i<n; i++){
		// ptcl_h[i] = Particle(x[i], m[i]);
		ptcl[i] = Particle(x[i], m[i]);
	}
	for(int i=n; i<ntg; i++){
		// ptcl_h[i] = Particle(0);
		ptcl[i] = Particle(0);
	}

	// hipMemcpy(ptcl_d, ptcl_h, ng * sizeof(Particle), hipMemcpyHostToDevice);
	ptcl.htod(ntg);
	
	dim3 grid(ng/NTHREAD, 1, 1);
	dim3 threads(NTHREAD, 1, 1);
	int sharedMemSize = NTHREAD * sizeof(Particle);
	// pot_kernel <<<grid, threads, sharedMemSize >>> (n, ptcl_d, phi_d);
	pot_kernel <<<grid, threads, sharedMemSize >>> (n, istart, ptcl, phi);

	// hipMemcpy(phi_h, phi_d, n * sizeof(float2), hipMemcpyDeviceToHost);
	phi.dtoh(ni);
	for(int i=0; i<ni; i++){
		// pot[i] = (double)phi_h[i].x + (double)phi_h[i].y;
		pot[i] = (double)phi[i].x + (double)phi[i].y;
	}

	phi.free();
	ptcl.free();
	double t1 = get_wtime();
#ifdef PROFILE
//  R: rank; Ni: input i particle; NTOT: total j particle; pot: calculation time
	fprintf(stderr, "[R.%d GPU Pot.A] Ni %d  NTOT %d  pot(s) %f\n",*rank,ni,n,t1 - t0);
#endif
}

extern "C"{
	void gpupot_(
            int *irank,
            int *istart,
            int *ni,
			int *n,
			double m[],
			double x[][3],
			double pot[]){
      gpupot(irank, *istart, *ni, *n, m, x, pot);
	}
}

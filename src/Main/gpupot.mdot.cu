#include "hip/hip_runtime.h"
//#include <iostream>
#include <cstdio>
// #include <cutil.h>
#include <cassert>
#ifdef CUDA_5
#  include <hip/hip_runtime_api.h>
#  define CUDA_SAFE_CALL checkCudaErrors
#else
#  include <cutil.h>
#endif
#include "cuda_pointer.h"
#include "cuda_share.h"

#define NTHREAD 64 // 64, 96, 128 or 192; should be same as the one in gpunb.gpu.cu
#define NJBLOCK 28 // 8800GTS/512 has 16
#define NIBLOCK 32 // 16 or 32 
#define NIMAX (NTHREAD * NIBLOCK) // 2048

#define NXREDUCE 32 // must be >NJBLOCK
#define NYREDUCE 8

//#define NAN_CHECK(val) assert((val) == (val));

//from gpunb.gpu.cu=================================//

struct Particle{
	float2 pos[3];
	float mass;
	float pad;

	Particle(double x[3], double m){
		pos[0] = float2_split(x[0]);
		pos[1] = float2_split(x[1]);
		pos[2] = float2_split(x[2]);
		mass = (float)m;

        NAN_CHECK(x[0]);
        NAN_CHECK(x[1]);
        NAN_CHECK(x[2]);
        NAN_CHECK(m);
	}
	Particle(int){
		pos[0].x = pos[0].y = pos[1].x = pos[1].y = pos[2].x = pos[2].y = mass = pad = 0.f;
	}
	__device__ Particle() {}
};

__global__ void pot_reduce_kernel(
		const int ni,
		const float2 phipart[][NJBLOCK],
        float2 phi[]){
  //thread x * y + block x============================//
  //thread x for NJBLOCK==============================//
	const int xid = threadIdx.x;
	const int yid = threadIdx.y;
	const int bid = blockIdx.x;
    //thread y & block x for active particle============//
	const int iaddr = yid + blockDim.y * bid;

	__shared__ float2 phishare[NYREDUCE][NXREDUCE];

    __syncthreads();
	if(xid < NJBLOCK){
      phishare[yid][xid] = phipart[iaddr][xid];
	}else{
      phishare[yid][xid] = make_float2(0.f,0.f);
	}
    __syncthreads();
	float2 *phis = phishare[yid];
    
#if NXREDUCE==32
	if(xid < 16) phis[xid] = float2_add(phis[xid],phis[xid + 16]);
#endif
	if(xid < 8) phis[xid] = float2_add(phis[xid],phis[xid + 8]);
	if(xid < 4) phis[xid] = float2_add(phis[xid],phis[xid + 4]);
	if(xid < 2) phis[xid] = float2_add(phis[xid],phis[xid + 2]);
	if(xid < 1) phis[xid] = float2_add(phis[xid],phis[xid + 1]);
	
	if(iaddr < ni){
      phi[iaddr] = float2_regularize(phis[0]);
	}
}

__global__ void pot_kernel_float(
          int ni,                           
          int n,
          Jparticle *ipbuf,
          Jparticle *ptcl,
          float2 phipart[][NJBLOCK]){
	int i = NTHREAD * blockIdx.x + threadIdx.x;
    int jbid = blockIdx.y;
    int jstart = (n * (jbid  )) / NJBLOCK;
    int jend   = (n * (jbid+1)) / NJBLOCK;

    Jparticle ip=Jparticle();
    if(i<ni) ip = ipbuf[i];
	float2 phii = make_float2(0.f, 0.f);
	for(int j=jstart; j<jend; j+= NTHREAD){
      __shared__ Jparticle jpbuf_f[NTHREAD];

      __syncthreads();
      float4 *src = (float4 *)&ptcl[j];
      float4 *dst = (float4 *)jpbuf_f;
      dst[threadIdx.x]         = src[threadIdx.x];
      dst[NTHREAD+threadIdx.x] = src[NTHREAD+threadIdx.x];
      __syncthreads();

      if(jend-j < NTHREAD) {
#pragma unroll 4        
        for(int jj=0; jj<jend-j; jj++){
			// if(j+jj == i) continue;
            Jparticle &jp = jpbuf_f[jj];
			float dx = jp.pos.x - ip.pos.x;
			float dy = jp.pos.y - ip.pos.y;
			float dz = jp.pos.z - ip.pos.z;
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }else{
#pragma unroll 8
        for(int jj=0; jj<NTHREAD; jj++){
			// if(j+jj == i) continue;
			Jparticle &jp = jpbuf_f[jj];
			float dx = jp.pos.x - ip.pos.x;
			float dy = jp.pos.y - ip.pos.y;
			float dz = jp.pos.z - ip.pos.z;
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }
      phii = float2_regularize(phii);
	}
#ifdef HIGH_CORR
    int istart = (ni* (jbid ))  / NJBLOCK;
    int iend   = (ni* (jbid+1)) / NJBLOCK;

    for(int j=istart; j<iend; j+= NTHREAD){
      __shared__ Jparticle jpbuf_f[NTHREAD];

      __syncthreads();
      float4 *src = (float4 *)&ipbuf[j];
      float4 *dst = (float4 *)jpbuf_f;
      dst[threadIdx.x]         = src[threadIdx.x];
      dst[NTHREAD+threadIdx.x] = src[NTHREAD+threadIdx.x];
      __syncthreads();

      if(iend-j < NTHREAD) {
#pragma unroll 4        
        for(int jj=0; jj<iend-j; jj++){
			// if(j+jj == i) continue;
            Jparticle &jp = jpbuf_f[jj];
			float dx = jp.pos.x - ip.pos.x;
			float dy = jp.pos.y - ip.pos.y;
			float dz = jp.pos.z - ip.pos.z;
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = -0.5 * jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }else{
#pragma unroll 8
        for(int jj=0; jj<NTHREAD; jj++){
			// if(j+jj == i) continue;
			Jparticle &jp = jpbuf_f[jj];
			float dx = jp.pos.x - ip.pos.x;
			float dy = jp.pos.y - ip.pos.y;
			float dz = jp.pos.z - ip.pos.z;
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = -0.5 * jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }
      phii = float2_regularize(phii);
	}
#endif
	phipart[i][jbid] = phii;
}

__global__ void pot_kernel(
          int ni,                           
          int n,
          Particle *ipbuf,
          Particle *ptcl,
          float2 phipart[][NJBLOCK]){
	int i = NTHREAD * blockIdx.x + threadIdx.x;
    int jbid = blockIdx.y;
    int jstart = (n * (jbid  )) / NJBLOCK;
    int jend   = (n * (jbid+1)) / NJBLOCK;

    Particle ip=Particle();
    if(i<ni) ip = ipbuf[i];
	float2 phii = make_float2(0.f, 0.f);
	for(int j=jstart; j<jend; j+= NTHREAD){
      __shared__ Particle jpbuf[NTHREAD];

      __syncthreads();
      float4 *src = (float4 *)&ptcl[j];
      float4 *dst = (float4 *)jpbuf;
      dst[threadIdx.x]         = src[threadIdx.x];
      dst[NTHREAD+threadIdx.x] = src[NTHREAD+threadIdx.x];
      __syncthreads();

      if(jend-j < NTHREAD) {
#pragma unroll 4        
        for(int jj=0; jj<jend-j; jj++){
			// if(j+jj == i) continue;
			Particle &jp = jpbuf[jj];
			float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
			float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
			float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y);
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }else{
#pragma unroll 8
        for(int jj=0; jj<NTHREAD; jj++){
			// if(j+jj == i) continue;
			Particle &jp = jpbuf[jj];
			float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
			float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
			float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y);
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }
      phii = float2_regularize(phii);
	}
#ifdef HIGH_CORR
    int istart = (ni* (jbid ))  / NJBLOCK;
    int iend   = (ni* (jbid+1)) / NJBLOCK;

    for(int j=istart; j<iend; j+= NTHREAD){
      __shared__ Particle jpbuf_f[NTHREAD];

      __syncthreads();
      float4 *src = (float4 *)&ipbuf[j];
      float4 *dst = (float4 *)jpbuf_f;
      dst[threadIdx.x]         = src[threadIdx.x];
      dst[NTHREAD+threadIdx.x] = src[NTHREAD+threadIdx.x];
      __syncthreads();

      if(iend-j < NTHREAD) {
#pragma unroll 4        
        for(int jj=0; jj<iend-j; jj++){
			// if(j+jj == i) continue;
            Particle &jp = jpbuf_f[jj];
			float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
			float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
			float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y);
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = 0.5 * jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }else{
#pragma unroll 8
        for(int jj=0; jj<NTHREAD; jj++){
			// if(j+jj == i) continue;
			Particle &jp = jpbuf_f[jj];
			float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
			float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
			float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y);
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = 0.5 * jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
      }
      phii = float2_regularize(phii);
	}
#endif
	phipart[i][jbid] = phii;
}

extern "C"  void gpunb_devinit_(int *irank);
//extern "C"  void gpunb_share_jp_(Jparticle *jpoint);
extern cudaPointer <Jparticle> jpbuf;
static cudaPointer <Jparticle> ipbuf;
static cudaPointer <float2> phi;
static cudaPointer <float2[NJBLOCK]> phipart;
static cudaPointer <Particle> ptcl;
static cudaPointer <Particle> ibuf;
static double tsend,tcalc;
static int icall,ini;

void init_ni() {
  phi.allocate(NIMAX);
  phipart.allocate(NIMAX);
}

void close_ni() {
  phi.free();
  phipart.free();
}  

void gpupot_init_float(int *irank){
  gpunb_devinit_(irank);
  ipbuf.allocate(NIMAX);
  init_ni();
  tsend=0.;
  tcalc=0.;
  icall=0;
  ini=0;
}

void gpupot_close_float(){
  close_ni();
  ipbuf.free();
}

void gpupot_init(int *irank, int n){
  int ntg = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));
  gpunb_devinit_(irank);
  ibuf.allocate(NIMAX);
  init_ni();
  ptcl.allocate(ntg);
  tsend=0.;
  tcalc=0.;
  icall=0;
  ini=0;
}

void gpupot_close(){
  ibuf.free();
  ptcl.free();
  close_ni();
}

void gpupot_float(
        int *rank,
        int ni,
		int n,
        int ishift,
        int list[],
        double dm[],
		double pot[]){

	gpunb_devinit_(rank);
    assert(ni<=NIMAX);
    //    Jparticle *jpoint=NULL;
    // gpunb_share_jp_(jpoint);
    //    assert(jpoint);
    assert(jpbuf.size);
    
	tcalc -= get_wtime();
    icall++;
    ini +=ni;
    //    cudaPointer <int> plist;
    int ng = NTHREAD * (ni/NTHREAD + (ni%NTHREAD ? 1 : 0));

    // ipbuf.allocate(ng);
	// phi.allocate(ng);
    // phipart.allocate(ng);
    //    plist.allocate(ni);
    // int *plist;
    // CUDA_SAFE_CALL(hipMalloc((void**)&plist, ni*sizeof(int)));
    // CUDA_SAFE_CALL(hipMemcpy(plist,list, ni*sizeof(int), hipMemcpyHostToDevice));
    // for(int i=0; i<ni; i++) {
    //   plist[i] = list[i] - 1;
    // }
    for(int i=0; i<ni; i++) {
      ipbuf[i] = jpbuf[list[i]-1+ishift];
      ipbuf[i].mass = dm[i];
    }
    for(int i=ni; i<ng; i++) {
      ipbuf[i] = Jparticle(0);
    }

    ipbuf.htod(ng);
    // plist.htod(ni);
	dim3 grid(ng/NTHREAD, NJBLOCK, 1);
	dim3 threads(NTHREAD, 1, 1);
	pot_kernel_float <<<grid, threads>>> (ni, n, ipbuf, jpbuf, phipart);

    const int ni8 = 1 + (ni-1) / NYREDUCE;
    dim3 rgrid (ni8, 1, 1);
    dim3 rthreads(NXREDUCE, NYREDUCE, 1);
    pot_reduce_kernel <<< rgrid, rthreads >>> (ni, phipart, phi);

    phi.dtoh(ni);

    for(int i=0; i<ni; i++){
      pot[i] = (double)phi[i].x + (double)phi[i].y;
	}

	tcalc += get_wtime();
// #ifdef PROFILE
// 	fprintf(stderr, "GPU potential (correction, float) - rank %d; Ni: %d; NTOT: %d; Time: %f sec\n",*rank,ni,n,t1 - t0);
// #endif
}

void gpupot_send(
        int *rank,
		int n,
		double m[],
		double x[][3]){
  	gpunb_devinit_(rank);
    tsend -= get_wtime();
	int ntg = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));
	for(int i=0; i<n; i++){
      ptcl[i] = Particle(x[i], m[i]);
	}
    for(int i=n; i<ntg; i++){
      ptcl[i] = Particle(0);
    }
	ptcl.htod(ntg);
	tsend += get_wtime();
// #ifdef PROFILE
// 	fprintf(stderr, "GPU potential send - rank %d; NTOT: %d; Time: %f sec\n",*rank,n,t1 - t0);
// #endif
}    

void gpupot(
        int *rank,
        int ni,
		int n,
        int ishift,
        int list[],
        double dm[],
		double pot[]){
	gpunb_devinit_(rank);

    //    assert(ni<=NIMAX);
    //DEBUG=============================================//
    //    printf("ni %d n %d list[0] %d m %lf\n",ni,n,list[0],m[0]);
	tcalc -= get_wtime();
    icall++;
    ini +=ni;
	// cudaPointer <float2> phi;
    // cudaPointer <float2[NJBLOCK]> phipart;
	// cudaPointer <Particle> ptcl;
    // cudaPointer <Particle> ipbuf;
    //    cudaPointer <int> plist;
    int ng = NTHREAD * (ni/NTHREAD + (ni%NTHREAD ? 1 : 0));
    //	int ntg = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));

    // ibuf.allocate(ng);
	// phi.allocate(ng);
    // phipart.allocate(ng);
	// ptcl.allocate(ntg);
    //    plist.allocate(ni);

    //  std::cout << n << " " << ng << " "<< ntg << std::endl;
    // for(int i=0; i<ni; i++) {
    //   plist[i] = list[i] - 1;
    // }
    for(int i=0; i<ni; i++) {
      ibuf[i] = ptcl[list[i]-1+ishift];
      ibuf[i].mass = dm[i];
    }
    for(int i=ni; i<ng; i++) {
      ibuf[i] = Particle(0);
    }
    
    //    printf("plist[0] %d\n",plist[0]);
    
    ibuf.htod(ng);
    //    plist.htod(ni);
    //    ptcl.dtoh(ntg);
	dim3 grid(ng/NTHREAD, NJBLOCK, 1);
	dim3 threads(NTHREAD, 1, 1);
    //	int sharedMemSize = NTHREAD * sizeof(Particle);
	pot_kernel <<<grid, threads>>> (ni, n, ibuf, ptcl, phipart);

    //    ptcl.dtoh(ni);
    
    const int ni8 = 1 + (ni-1) / NYREDUCE;
    dim3 rgrid (ni8, 1, 1);
    dim3 rthreads(NXREDUCE, NYREDUCE, 1);
    pot_reduce_kernel <<< rgrid, rthreads >>> (ni, phipart, phi);

    phi.dtoh(ni);

    for(int i=0; i<ni; i++){
      pot[i] = (double)phi[i].x + (double)phi[i].y;
	}

	// phi.free();
    // phipart.free();
	// ptcl.free();
    // ipbuf.free();
	tcalc += get_wtime();
// #ifdef PROFILE
// 	fprintf(stderr, "GPU potential (correction) - rank %d; Ni: %d; NTOT: %d; Time: %f sec\n",*rank,ni,n,t1 - t0);
// #endif
}

void gpupot_profile(int irank) {
#ifdef PROFILE
  if(icall) {
    // R.: rank;
    // Ncall: number of call gpupot and gpupot_float during two checking time(adjust time interval)
    // <Ni>: averaged i particles per call;
    // send: j particle sending time;
    // pot:  potential calculation time;
    fprintf(stderr, "[R.%d GPU Pot.C] Ncall %d  <Ni> %d   send(s) %f (ave) %f  pot(s) %f (ave) %f\n",irank,icall,ini/icall,tsend,tsend/icall,tcalc,tcalc/icall);
  }
  ini = icall = 0;
  tsend = tcalc = 0.;
#else
  return;
#endif
}

extern "C"{
  void gpupot_init_(
             int *irank,
             int *n){
    gpupot_init(irank,*n);
  }
  void gpupot_close_(){
    gpupot_close();
  }
  void gpupot_send_(int *irank, int *n, double m[], double x[][3]) {
    gpupot_send(irank,*n,m,x);
  }
  void gpupot_dm_(
            int *irank,
            int *ni,
			int *n,
            int *ishift,
            int list[],
            double dm[],
			double pot[]){
    gpupot(irank, *ni, *n, *ishift, list, dm, pot);
  }
  void gpupot_init_float_(
           int *irank ){
    gpupot_init_float(irank);
  }
  void gpupot_close_float_(){
    gpupot_close_float();
  }
  void gpupot_float_(
            int *irank,
            int *ni,
			int *n,
            int *ishift,
            int list[],
            double dm[],
			double pot[]){
    gpupot_float(irank, *ni, *n, *ishift, list, dm, pot);
  }
  void gpupot_mdot_profile_(int *irank){
    gpupot_profile(*irank);
  }
}
